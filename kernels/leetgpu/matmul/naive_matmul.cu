#include <hip/hip_runtime.h>

__global__ void matrix_multiplication_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < K) {
        float pValue = 0;
        for (int n = 0; n < N; n++) {
            pValue += A[row * N + n] * B[n * K + col];
        }
        C[row * K + col] = pValue;
    }
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* A, const float* B, float* C, int M, int N, int K) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((K + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}
